#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <errno.h>
#include "../constants.h"
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

char seq1_list[MAX_PAIRS][MAX_SEQ_LENGTH];
char seq2_list[MAX_PAIRS][MAX_SEQ_LENGTH];
int score_matrix[MAX_PAIRS];

int load_sequences(const char *filename)
{
    FILE *file = fopen(filename, "r");
    if ( file == NULL ) {
        perror("File Open Error");
        printf("Error opening file %s: %d", filename, errno);
        exit(1);
    }

    int count = 0;

    while (fscanf( file, "%200[^,],%200s\n", seq1_list[count], seq2_list[count]) == 2) {
        count++;
    }

    fclose(file);
    return count;
}

__global__ void smith_waterman_kernel(char* d_seq1, char* d_seq2, int* d_offsets, int* d_scores)    
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int offset = d_offsets[idx];
    char* s1_pointer = &d_seq1[offset * MAX_SEQ_LENGTH];
    char* s2_pointer = &d_seq2[offset * MAX_SEQ_LENGTH];

    int len1 = MAX_SEQ_LENGTH;
    int len2 = MAX_SEQ_LENGTH;

    // High Scoring Local Alignment Matrix (H)
    int H[MAX_SEQ_LENGTH][MAX_SEQ_LENGTH] = {0};

    int score_diagonal, score_up, score_left, max_score = 0;

    for (int i = 1; i <= len1 ; i++)
    {
        for (int j = 1; j <= len2 ; j++)
        {
            score_diagonal = H[i - 1][j - 1] + (s1_pointer[i - 1] == s2_pointer[j - 1] ? MATCHING_SCORE : MISMATCHING_SCORE);
            score_up = H[i - 1][j] + GAP_PENALTY;
            score_left = H[i][j - 1] + GAP_PENALTY;
            H[i][j] = max(0, max(score_diagonal, max(score_up, score_left)));

            if (H[i][j] > max_score) 
                max_score = H[i][j];
        }
    }

    d_scores[idx] = max_score;
}

void save_score_matrix(const char *filename)
{
    FILE *file = fopen(filename, "w");
    if (file == NULL)
    {
        perror("File Open Error");
        printf("Error opening file %s: %d\n", filename, errno);
        exit(1);
    }

    for(int i = 0; i < MAX_PAIRS; i++)
    {
        fprintf(file, "Index %d, Score: %d\n", i, score_matrix[i]);
    }

    fclose(file);
    printf("Score matrix saved to %s\n", filename);
}

int main()
{
    printf("\n"
           "========================================\n"
           "Smith-Waterman Algorithm - Using CUDA\n"
           "========================================\n");
    int n = load_sequences("data/DNASequences.txt");
    printf("Loaded %d pairs of sequences.\n", n);

    char *h_seq1 = (char*) malloc(n * MAX_SEQ_LENGTH * sizeof(char));
    char *h_seq2 = (char*) malloc(n * MAX_SEQ_LENGTH * sizeof(char));
    int *h_offsets = (int*) malloc(n * sizeof(int));
    int *h_scores = (int*) malloc(n * sizeof(int));

    char *d_seq1, *d_seq2;
    int *d_offsets, *d_scores;
    hipMalloc(&d_seq1, n * MAX_SEQ_LENGTH * sizeof(char));
    hipMalloc(&d_seq2, n * MAX_SEQ_LENGTH * sizeof(char));
    hipMalloc(&d_offsets, n * sizeof(int));
    hipMalloc(&d_scores, n * sizeof(int));

    for ( int i = 0; i < n; i++ )
    {
        memcpy(&h_seq1[i * MAX_SEQ_LENGTH], seq1_list[i], MAX_SEQ_LENGTH);
        memcpy(&h_seq2[i * MAX_SEQ_LENGTH], seq2_list[i], MAX_SEQ_LENGTH);
        h_offsets[i] = i;
    }

    hipMemcpy(d_seq1, h_seq1, n * MAX_SEQ_LENGTH, hipMemcpyHostToDevice);
    hipMemcpy(d_seq2, h_seq2, n * MAX_SEQ_LENGTH, hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, h_offsets, n * sizeof(int), hipMemcpyHostToDevice);

    int threads_per_block = 256;
    int num_blocks = (n + threads_per_block - 1) / threads_per_block;

    struct timeval start, end;
    gettimeofday(&start, NULL);

    smith_waterman_kernel<<<num_blocks, threads_per_block>>>(d_seq1, d_seq2, d_offsets, d_scores);

    gettimeofday(&end, NULL);

    printf("Completed alignment of %d sequence pairs.\n", n);
    
    long start_time = (start.tv_sec * 1000000 + start.tv_usec);
    long end_time = (end.tv_sec * 1000000 + end.tv_usec);
    long elapsed_time = end_time - start_time;
    printf("Total time taken: %0.6f seconds\n", (float)elapsed_time / 1000000);

    hipMemcpy(h_scores, d_scores, n * sizeof(int), hipMemcpyDeviceToHost);

    memcpy(score_matrix, h_scores, n * sizeof(int));

    hipFree(d_seq1);
    hipFree(d_seq2);
    hipFree(d_offsets);
    hipFree(d_scores);

    free(h_seq1);
    free(h_seq2);
    free(h_offsets);
    free(h_scores);

    save_score_matrix("output/phase02_code_output_max_scores.txt");
}